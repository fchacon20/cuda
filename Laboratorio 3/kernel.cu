#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

// consideraciones

/*
Se trabajar�a con una matriz cuadrada de 10^8 elementos, i.e. N = M = 10^4

El tamano de un bloque de hebras siempre sera 256.

Los valores especificados en los dos puntos anteriores pueden ser declarados como constantes en
tiempo de compilacion.

La matriz A y el vector x pueden ser inicializados con los valores que guste mientras no contengan
valores nulos (0). Como consejo, se le recomienda que todos los valores sean 1 para entonces poder
comprobar si el resultado es correcto cuando todos los valores en b sean 10^4.
*/

__global__ void kernelA(int *A, int*x, int*b, int N) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;

	int i = tId % N;
	int j = tId / N;
	if (i < N && j < N) {
		//atomicAdd(A[i][j] * x[j], b[i]);
	}
	/*
	Este kernel utiliza N x N = 10^8 hebras. Cada hebra esta asociada a un elemento a_i,j de la matriz A
	multiplic�ndolo por el valor x_j correspondiente y sumando este resultado al elemento en la i-�sima
	posici�n del vector b.
	*/
}

__global__ void kernelx(int *A, int*x, int*b, int N) {
	/*
	Este kernel utiliza N = 10^4 hebras. Cada hebra esta asociada a un elemento x_j del vector x,
	sumando a cada uno de los N valores b_i	, la multiplicacion de dicho x_j
	por el correspondiente elemento a_i,j
	*/

	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < N) {
		for (int j = 0; j < N; j++) {
			int mult = x[tId] * A[j];
			atomicAdd(&b[tId], mult);
		}
	}
}


__global__ void kernelb(int *A, int*x, int*b, int N) {


}

__global__ void kernelRed(int *A, int*x, int*b, int N) {
}

__global__ void kernelSM(int *A, int*x, int*b, int N) {
}

__global__ void kernelCM(int *A, int*b, int N) {
}

void fillArray(int *a, int n) {
	for (int i = 0; i < n; ++i)
		a[i] = 1;
}

int main() {

	int N = 1e1;
	int M = 1e1;
	int* A = new int[N*M];
	int* X = new int[N];
	int* B = new int[N];
	
	int *devA, *devX, *devB;
	hipMalloc(&devA, N*M * sizeof(int));
	hipMalloc(&devX, N * sizeof(int));
	hipMalloc(&devB, N * sizeof(int));

	fillArray(A, N*M);
	fillArray(X, N);
	
	hipMemcpy(devA, A, N*M * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devX, X, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(devB, 0, N * sizeof(int));

	int block_size = 256;

	// KernelX
	int grid_size = (int)ceil((float)N / block_size);

	kernelx << <grid_size, block_size >> > (devA, devX, devB, N);

	hipMemcpy(B, devB, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		cout << B[i] << endl;
	}
	

	return 0;
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>

#define E 2.71828182845904523536

using namespace std;

void actualizadorCPU(float * soluciones, int m, float deltaT, float tI) {
	for (int j = 0; j < m; j++)
		soluciones[j] = soluciones[j] + deltaT * (4 * (tI - deltaT) - soluciones[j] + 3 + j);
}

__global__ void actualizadorGPU(float * devSoluciones, int m, float deltaT, float tI) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if (tId < m) {
		devSoluciones[tId] = devSoluciones[tId] + deltaT * (4 * (tI - deltaT) - devSoluciones[tId] + 3 + tId);
	}
}

void secondCPU_GPU(int m) {
	int n = 1000;
	long duration;
	float deltaT = 0.001;
	clock_t t1, t2;

	float * soluciones = new float[m];
	for (int j = 0; j < m; j++)
		soluciones[j] = j;

	float * devSoluciones;
	hipMalloc(&devSoluciones, m * sizeof(float));
	hipMemcpy(devSoluciones, soluciones, n * sizeof(float), hipMemcpyHostToDevice);

	t1 = clock();
	for (int i = 1; i < n; i++)
		actualizadorCPU(soluciones, m, deltaT, deltaT*i);
	t2 = clock();
	duration = 1000 * (double)(t2 - t1) / CLOCKS_PER_SEC;
	cout << "[CPU] Tama�o m = " << m << ": " << duration << " [ms]" << endl;

	hipEvent_t ct1, ct2;
	float dt;
	int grid_size = (int)ceil((float)m / 256);
	int block_size(256);
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);
	for (int i = 1; i < n; i++)
		actualizadorGPU << <grid_size, block_size >> > (devSoluciones, m, deltaT, deltaT*i);
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);
	hipMemcpy(soluciones, devSoluciones, m * sizeof(float), hipMemcpyDeviceToHost);
	cout << "[GPU] Tama�o m = : " << m << ": " << dt << " [ms]" << endl;

	hipFree(devSoluciones);
	delete soluciones;
}

void fixedM() {
	int n = 1000;
	int m = 1e8;
	float deltaT = 0.001;
	int threads[4] = { 64, 128, 256, 512 };

	float * soluciones = new float[m];
	for (int j = 0; j < m; j++)
		soluciones[j] = j;

	float * devSoluciones;
	hipMalloc(&devSoluciones, m * sizeof(float));

	hipEvent_t ct1, ct2;
	float dt;
	for (int t = 0; t < 4; t++) {
		hipMemcpy(devSoluciones, soluciones, m * sizeof(float), hipMemcpyHostToDevice);
		int grid_size = (int)ceil((float)m / threads[t]);
		int block_size(threads[t]);
		hipEventCreate(&ct1);
		hipEventCreate(&ct2);
		hipEventRecord(ct1);
		for (int i = 1; i < n; i++)
			actualizadorGPU << <grid_size, block_size >> > (devSoluciones, m, deltaT, deltaT*i);
		hipEventRecord(ct2);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);
		hipMemcpy(soluciones, devSoluciones, m * sizeof(float), hipMemcpyDeviceToHost);
		cout << "[GPU] Hebras = : " << threads[t] << ": " << dt << " [ms]" << endl;
	}

	hipFree(devSoluciones);
	delete soluciones;
}

int main() {

	//secondCPU_GPU(10000);
	//secondCPU_GPU(100000);
	//secondCPU_GPU(1000000);
	//secondCPU_GPU(10000000);
	//secondCPU_GPU(100000000);
	fixedM();

	return 0;
}
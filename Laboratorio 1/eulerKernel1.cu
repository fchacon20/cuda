#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <ctime>
#include <iomanip>

#define E 2.71828182845904523536

using namespace std;

void __global__ calculoGPU(float t0, float y0, float deltaT, int n, float * soluciones_GPU) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	int i;
	float sum = 0;
	if (tId == 0)
		soluciones_GPU[0] = y0;
	else if (tId < n) {
		for (i = 1; i < tId; i++)
			sum = sum + powf(E, -1 * (i - 1) * deltaT);
		soluciones_GPU[tId] = (sum * deltaT) + y0;
	}
}

void eulerMethodGPU(float t0, float y0, float deltaT) {
	float tN = 10;
	hipEvent_t ct1, ct2;
	float dt;
	int n = (int)((tN - t0) / deltaT);
	int block_size = 256;
	int grid_size = (int)ceil((float)n / block_size);
	float * soluciones = new float[n];
	float * soluciones_GPU;
	hipMalloc(&soluciones_GPU, n * sizeof(float));
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);
	calculoGPU << <grid_size, block_size >> > (t0, y0, deltaT, n, soluciones_GPU);
	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);
	hipMemcpy(soluciones, soluciones_GPU, n * sizeof(float), hipMemcpyDeviceToHost);

	cout << "[GPU] DeltaT = " << deltaT << ": " << dt << " [ms]" << endl;

	hipFree(soluciones_GPU);
	delete[] soluciones;
}

float* eulerMethodCPU(int t0, float y0, float deltaT) {
	int n = int(10 / deltaT);
	float* y = new float[n];
	float sum = 0.0;

	y[0] = y0;
	for (int i = 1; i < n; i++) {
		sum = sum + powf((float) E, -1 * (i - 1)*deltaT);
		y[i] = y0 + deltaT * sum;
	}
	return y;
}


int main() {

	float* y;
	float deltaT[6] = { (float) 1e-1, (float) 1e-2, (float) 1e-3,
						(float) 1e-4, (float) 1e-5, (float) 1e-6};
	clock_t t1;
	clock_t t2;
	double duration;

	for (int i = 0; i < 6; i++) {
		t1 = clock();
		y = eulerMethodCPU(0, -1, deltaT[i]);
		t2 = clock();
		duration = 1000 * (double)(t2 - t1) / CLOCKS_PER_SEC;
		cout << "[CPU] DeltaT = " << deltaT[i] << ": " << duration << " [ms]" << endl;
	}

	cout << "---------------" << endl;
	eulerMethodGPU(0, -1, deltaT[0]);
	eulerMethodGPU(0, -1, deltaT[1]);
	eulerMethodGPU(0, -1, deltaT[2]);
	eulerMethodGPU(0, -1, deltaT[3]);

	delete[] y;
	return 0;
}
#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>

#define E 2.71828182845904523536

using namespace std;

void __global__ calculoGPU(float t0, float y0, float deltaT, int n, float * soluciones_GPU){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int i;
    float sum = 0;
    if(tId == 0)
        soluciones_GPU[0] = y0;
    else if(tId < n){
        for(i = 1; i < tId; i++)
            sum = sum + powf(E, -1 * (i - 1) * deltaT);
        soluciones_GPU[tId] = (sum * deltaT) + y0;
    }
}

void eulerMethodGPU(float t0, float y0, float deltaT){
    FILE * archivo;
    float tN = 10;
    int i;
    hipEvent_t ct1, ct2;
    float dt;
    int n = (tN - t0)/deltaT;
    int block_size = 256;
    int grid_size = (int)ceil((float)n / block_size);
    float * soluciones = new float[n];
    float * soluciones_GPU;
    hipMalloc(&soluciones_GPU, n * sizeof(float));
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    calculoGPU << <grid_size, block_size>> >(t0, y0, deltaT, n, soluciones_GPU);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
    hipMemcpy(soluciones, soluciones_GPU, n * sizeof(float), hipMemcpyDeviceToHost);

	cout << "[GPU] DeltaT = " << deltaT << ": " << dt << " [ms]" << endl;

    archivo = fopen("eulerGPU.txt", "a");
    for(i = 0; i < n; i++){
        if(i == n-1)
            fprintf(archivo, "%f\n", soluciones[i]);
        else
            fprintf(archivo, "%f ", soluciones[i]);
    }
    fclose(archivo);
    hipFree(soluciones_GPU);
    delete soluciones;
}

float* eulerMethodCPU(int t0, float y0, float deltaT) {
	int n = int(10 / deltaT);
	float* y = new float[n];
	float sum = 0.0;
	
	y[0] = y0;
	for (int i = 1; i < n; i++)	{
		sum = sum + powf(E, -1 *(i-1)*deltaT);
		y[i] = y0 + deltaT * sum;
	}
	return y;
}


int main() {
	
	float* y;
	float deltaT[6] = { 0.1, 0.01, 0.001, 0.0001, 0.00001, 0.000001 };
	clock_t t1;
	clock_t t2;
	long duration;

	ofstream outputFile;
	outputFile.open("euler.txt");

	for (int i = 0; i < 6; i++) {
		t1 = clock();
		y = eulerMethodCPU(0, -1, deltaT[i]);
		t2 = clock();
		duration = 1000*(double)(t2 - t1) / CLOCKS_PER_SEC;
		cout << "[CPU] DeltaT = " << deltaT[i] << ": " << duration << " [ms]" << endl;

		for (int j = 0; j < int(10 / deltaT[i])-1; j++)
			outputFile << y[j] << " ";
		outputFile << y[int(10 / deltaT[i]) - 1] << endl;
	}	
	cout << "---------------" << endl;
	eulerMethodGPU(0, -1, 0.1);
    eulerMethodGPU(0, -1, 0.01);
    eulerMethodGPU(0, -1, 0.001);
	eulerMethodGPU(0, -1, 0.0001);
	
	outputFile.close();
	delete[] y;
	return 0;
}
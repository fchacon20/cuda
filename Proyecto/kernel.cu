#include "hip/hip_runtime.h"
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace std;

void initialPoints(float *x, float *y, int M, int a, int b) {
	for (int i = 1; i <= M; ++i) {
		x[i - 1] = (double)(a + b) / 2 + (double)(((b - a) / 2.0)*cos((2.0*i - 1.0)*M_PI / ((double)2.0*M)));
		y[i - 1] = cos(x[i - 1]);
	}
}

void generateX(float *x, int n, int a, int b) {
	for (int i = 1; i <= n; ++i) {
		x[i-1] = ((float)(b-a)/(n))*i;
	}
}

__global__ void lagrange_uno(const float * __restrict__ X, const float * __restrict__ Y,
							 const float * __restrict__ N_x, float* N_y, int N, int M) {
	int tId = threadIdx.x + blockIdx.x * blockDim.x;

	if (tId < N) {
		float sum = 0;
		float prod;

		for (int i = 0; i < M; i++) {
			prod = 1;
			if (N_x[tId] == X[i]) continue;
			for (int j = 0; j < M; j++) {
				if (j == i) continue;
				prod = prod * (N_x[tId] - X[j]) / (X[i] - X[j]);
			}
			sum = sum + prod * Y[i];
		}
		N_y[tId] = sum;
	}
}

int main() {
	int N = 1000000; // Cantidad de puntos a interpolar
	int M = 30; // Cantidad de puntos a utilizar de la funci�n original
	int a = 0;
	int b = 100;

	float *X, *Y;		// Arreglos conteniendo coordenadas X e Y de puntos de la funci�n original
	float *N_x, *N_y;	// Arreglos conteniendo coordenadas X e Y de puntos a interpolar.
						// N_x deben generarse, N_y deben calcularse usando el kernel

	float *x = new float[M]; // coordenadas x de la funci�n
	float *y = new float[M]; // coordenadas y de la funci�n
	float *x_generados = new float[N];
	float *y_generados = new float[N];
	
	initialPoints(x, y, M, a, b);
	generateX(x_generados, N, a, b);

	// Saving input
	//ofstream outfile("C:/Usuarios/Wil/Escritorio/Wil/initialPoints.txt");
	ofstream outfile("./initialPoints.txt");
	for (int i = 0; i < M - 1; ++i)
		outfile << x[i] << ",";
	outfile << x[M - 1] << "\n";
	for (int i = 0; i < M - 1; ++i)
		outfile << y[i] << ",";
	outfile << y[M - 1] << "\n";
	outfile.close();

	int block_size = 256;
	int grid_size = (int)ceil((float)N / block_size);

	hipEvent_t ct1, ct2;
	float dt;

	hipMalloc(&X, M * sizeof(float));
	hipMalloc(&Y, M * sizeof(float));
	hipMalloc(&N_x, N * sizeof(float));
	hipMalloc(&N_y, N * sizeof(float));

	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	hipMemcpy(X, x, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Y, y, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(N_x, x_generados, N * sizeof(float), hipMemcpyHostToDevice);

	lagrange_uno << < grid_size, block_size >> > (X, Y, N_x, N_y, N, M);
	hipMemcpy(y_generados, N_y, N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	cout << "[GPU] Duration: " << dt << " ms" << endl;

	ofstream outfile2("./output.txt");
	for (int i = 0; i < N - 1; ++i)
		outfile2 << x_generados[i] << ",";
	outfile2 << x_generados[N - 1] << "\n";
	for (int i = 0; i < N - 1; ++i)
		outfile2 << y_generados[i] << ",";
	outfile2 << y_generados[N - 1] << "\n";
	outfile2.close();

	hipFree(X);
	hipFree(Y);
	hipFree(N_x);
	hipFree(N_y);
	
	delete x;
	delete y;
	delete x_generados;
	delete y_generados;

	return 0;
}